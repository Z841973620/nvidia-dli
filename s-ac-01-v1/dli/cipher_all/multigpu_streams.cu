#include "hip/hip_runtime.h"
#include <cstdint>
#include <iostream>
#include "helpers.cuh"
#include "encryption.cuh"

void encrypt_cpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters, bool parallel=true) {

    #pragma omp parallel for if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
        data[entry] = permute64(entry, num_iters);
}

__global__ 
void decrypt_gpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters) {

    const uint64_t thrdID = blockIdx.x*blockDim.x+threadIdx.x;
    const uint64_t stride = blockDim.x*gridDim.x;

    for (uint64_t entry = thrdID; entry < num_entries; entry += stride)
        data[entry] = unpermute64(data[entry], num_iters);
}

bool check_result_cpu(uint64_t * data, uint64_t num_entries,
                      bool parallel=true) {

    uint64_t counter = 0;

    #pragma omp parallel for reduction(+: counter) if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
        counter += data[entry] == entry;

    return counter == num_entries;
}

int main (int argc, char * argv[]) {

    Timer timer;
    Timer overall;

    const uint64_t num_entries = 1UL << 26;
    const uint64_t num_iters = 1UL << 10;
    const bool openmp = true;

    const uint64_t num_gpus = 4;
    const uint64_t num_streams = 32;
    const uint64_t chunk_size = sdiv(sdiv(num_entries, num_gpus), num_streams);

    hipStream_t streams[num_gpus][num_streams];

    timer.start();
    for (uint64_t gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        for (uint64_t stream = 0; stream < num_streams; stream++)
            hipStreamCreate(&streams[gpu][stream]);
    }
    timer.stop("create streams");
    check_last_error();


    timer.start();
    uint64_t * data_cpu, * data_gpu[num_gpus];
    hipHostMalloc(&data_cpu, sizeof(uint64_t)*num_entries);
    for (uint64_t gpu = 0; gpu < num_gpus; gpu++) {

        hipSetDevice(gpu);

        const uint64_t lower = chunk_size*num_streams*gpu;
        const uint64_t upper = min(lower+chunk_size*num_streams, num_entries);
        const uint64_t width = upper-lower;

        hipMalloc(&data_gpu[gpu], sizeof(uint64_t)*width);
    }
    timer.stop("allocate memory");
    check_last_error();

    timer.start();
    encrypt_cpu(data_cpu, num_entries, num_iters, openmp);
    timer.stop("encrypt data on CPU");

    overall.start();
    timer.start();
    for (uint64_t gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        for (uint64_t stream = 0; stream < num_streams; stream++) {


            const uint64_t offset = chunk_size*stream;
            const uint64_t lower = chunk_size*num_streams*gpu+offset;
            const uint64_t upper = min(lower+chunk_size, num_entries);
            const uint64_t width = upper-lower;

            hipMemcpyAsync(data_gpu[gpu]+offset, data_cpu+lower, 
                            sizeof(uint64_t)*width, hipMemcpyHostToDevice,
                            streams[gpu][stream]);
    
            decrypt_gpu<<<80*32, 64, 0, streams[gpu][stream]>>>
                (data_gpu[gpu]+offset, width, num_iters);
    
            hipMemcpyAsync(data_cpu+lower, data_gpu[gpu]+offset, 
                            sizeof(uint64_t)*width, hipMemcpyDeviceToHost,
                            streams[gpu][stream]);
        }
    }

    for (uint64_t gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        for (uint64_t stream = 0; stream < num_streams; stream++) {
            hipStreamSynchronize(streams[gpu][stream]);
        }
    }
    timer.stop("asynchronous H2D -> kernel -> D2H multiGPU");
    overall.stop("total time on GPU");
    check_last_error();

    timer.start();
    const bool success = check_result_cpu(data_cpu, num_entries, openmp);
    std::cout << "STATUS: test " 
              << ( success ? "passed" : "failed")
              << std::endl;
    timer.stop("checking result on CPU");

    timer.start();
    for (uint64_t gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        for (uint64_t stream = 0; stream < num_streams; stream++) {
            hipStreamDestroy(streams[gpu][stream]);
        }
    }
    timer.stop("destroy streams");
    check_last_error();

    timer.start();
    hipHostFree(data_cpu);
    for (uint64_t gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        hipFree(data_gpu[gpu]);
    }    
    timer.stop("free memory");
    check_last_error();
}
