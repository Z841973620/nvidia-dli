#include "hip/hip_runtime.h"
#include <cstdint>
#include <iostream>
#include "helpers.cuh"
#include "encryption.cuh"

void encrypt_cpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters, bool parallel=true) {

    #pragma omp parallel for if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
        data[entry] = permute64(entry, num_iters);
}

__global__ 
void decrypt_gpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters) {

    const uint64_t thrdID = blockIdx.x*blockDim.x+threadIdx.x;
    const uint64_t stride = blockDim.x*gridDim.x;

    for (uint64_t entry = thrdID; entry < num_entries; entry += stride)
        data[entry] = unpermute64(data[entry], num_iters);
}

bool check_result_cpu(uint64_t * data, uint64_t num_entries,
                      bool parallel=true) {

    uint64_t counter = 0;

    #pragma omp parallel for reduction(+: counter) if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
        counter += data[entry] == entry;

    return counter == num_entries;
}

int main (int argc, char * argv[]) {

    Timer timer;
    Timer overall;

    const uint64_t num_entries = 1UL << 26;
    const uint64_t num_iters = 1UL << 10;
    const bool openmp = true;

    const uint64_t num_streams = 32;
    const uint64_t chunk_size = sdiv(num_entries, num_streams);

    timer.start();
    uint64_t * data_cpu, * data_gpu;
    hipHostMalloc(&data_cpu, sizeof(uint64_t)*num_entries);
    hipMalloc    (&data_gpu, sizeof(uint64_t)*num_entries);
    timer.stop("allocate memory");
    check_last_error();

    timer.start();
    encrypt_cpu(data_cpu, num_entries, num_iters, openmp);
    timer.stop("encrypt data on CPU");

    timer.start();
    hipStream_t streams[num_streams];
    for (uint64_t stream = 0; stream < num_streams; stream++)
        hipStreamCreate(&streams[stream]);
    timer.stop("create streams");
    check_last_error();

    overall.start();
    timer.start();
    for (uint64_t stream = 0; stream < num_streams; stream++) {
        
        const uint64_t lower = chunk_size*stream;
        const uint64_t upper = min(lower+chunk_size, num_entries);
        const uint64_t width = upper-lower;

        hipMemcpyAsync(data_gpu+lower, data_cpu+lower, 
               sizeof(uint64_t)*width, hipMemcpyHostToDevice, 
               streams[stream]);
    
        decrypt_gpu<<<80*32, 64, 0, streams[stream]>>>
            (data_gpu+lower, width, num_iters);

        hipMemcpyAsync(data_cpu+lower, data_gpu+lower, 
               sizeof(uint64_t)*width, hipMemcpyDeviceToHost, 
               streams[stream]);
    }    
    timer.stop("asynchronous H2D->kernel->D2H");
    overall.stop("total time on GPU");
    check_last_error();
    
    timer.start();
    const bool success = check_result_cpu(data_cpu, num_entries, openmp);
    std::cout << "STATUS: test " 
              << ( success ? "passed" : "failed")
              << std::endl;
    timer.stop("checking result on CPU");

    timer.start();
    for (uint64_t stream = 0; stream < num_streams; stream++)
        hipStreamDestroy(streams[stream]);    
    timer.stop("destroy streams");
    check_last_error();

    timer.start();
    hipHostFree(data_cpu);
    hipFree    (data_gpu);
    timer.stop("free memory");
    check_last_error();
}
