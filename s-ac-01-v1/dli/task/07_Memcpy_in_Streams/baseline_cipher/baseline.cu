#include "hip/hip_runtime.h"
#include <cstdint>
#include <iostream>
#include "helpers.cuh"
#include "encryption.cuh"

void encrypt_cpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters, bool parallel=true) {

    #pragma omp parallel for if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
        data[entry] = permute64(entry, num_iters);
}

__global__ 
void decrypt_gpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters) {

    const uint64_t thrdID = blockIdx.x*blockDim.x+threadIdx.x;
    const uint64_t stride = blockDim.x*gridDim.x;

    for (uint64_t entry = thrdID; entry < num_entries; entry += stride)
        data[entry] = unpermute64(data[entry], num_iters);
}

bool check_result_cpu(uint64_t * data, uint64_t num_entries,
                      bool parallel=true) {

    uint64_t counter = 0;

    #pragma omp parallel for reduction(+: counter) if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
        counter += data[entry] == entry;

    return counter == num_entries;
}

int main (int argc, char * argv[]) {

    const char * encrypted_file = "/dli/task/encrypted";

    Timer timer;

    const uint64_t num_entries = 1UL << 26;
    const uint64_t num_iters = 1UL << 10;
    const bool openmp = true;

    uint64_t * data_cpu, * data_gpu;
    hipHostMalloc(&data_cpu, sizeof(uint64_t)*num_entries);
    hipMalloc    (&data_gpu, sizeof(uint64_t)*num_entries);
    check_last_error();

    if (!encrypted_file_exists(encrypted_file)) {
        encrypt_cpu(data_cpu, num_entries, num_iters, openmp);
        write_encrypted_to_file(encrypted_file, data_cpu, sizeof(uint64_t)*num_entries);
    } else {
        read_encrypted_from_file(encrypted_file, data_cpu, sizeof(uint64_t)*num_entries);
    }

    timer.start();
    hipMemcpy(data_gpu, data_cpu, 
               sizeof(uint64_t)*num_entries, hipMemcpyHostToDevice);
    check_last_error();

    decrypt_gpu<<<80*32, 64>>>(data_gpu, num_entries, num_iters);
    check_last_error();

    hipMemcpy(data_cpu, data_gpu, 
               sizeof(uint64_t)*num_entries, hipMemcpyDeviceToHost);
    timer.stop("total time on GPU");
    check_last_error();

    const bool success = check_result_cpu(data_cpu, num_entries, openmp);
    std::cout << "STATUS: test " 
              << ( success ? "passed" : "failed")
              << std::endl;

    hipHostFree(data_cpu);
    hipFree    (data_gpu);
    check_last_error();
}
