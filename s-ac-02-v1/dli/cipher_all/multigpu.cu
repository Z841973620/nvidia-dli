#include "hip/hip_runtime.h"
#include <cstdint>
#include <iostream>
#include "helpers.cuh"
#include "encryption.cuh"

void encrypt_cpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters, bool parallel=true) {

    #pragma omp parallel for if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
        data[entry] = permute64(entry, num_iters);
}

__global__ 
void decrypt_gpu(uint64_t * data, uint64_t num_entries, 
                 uint64_t num_iters) {

    const uint64_t thrdID = blockIdx.x*blockDim.x+threadIdx.x;
    const uint64_t stride = blockDim.x*gridDim.x;

    for (uint64_t entry = thrdID; entry < num_entries; entry += stride)
        data[entry] = unpermute64(data[entry], num_iters);
}

bool check_result_cpu(uint64_t * data, uint64_t num_entries,
                      bool parallel=true) {

    uint64_t counter = 0;

    #pragma omp parallel for reduction(+: counter) if (parallel)
    for (uint64_t entry = 0; entry < num_entries; entry++)
        counter += data[entry] == entry;

    return counter == num_entries;
}

int main (int argc, char * argv[]) {

    Timer timer;
    Timer overall;

    const uint64_t num_entries = 1UL << 26;
    const uint64_t num_iters = 1UL << 10;
    const bool openmp = true;

    const uint64_t num_gpus = 4;
    const uint64_t chunk_size = sdiv(num_entries, num_gpus);

    timer.start();
    uint64_t * data_cpu, * data_gpu[num_gpus];
    hipHostMalloc(&data_cpu, sizeof(uint64_t)*num_entries);
    for (uint64_t gpu = 0; gpu < num_gpus; gpu++) {

        hipSetDevice(gpu);

        const uint64_t lower = chunk_size*gpu;
        const uint64_t upper = min(lower+chunk_size, num_entries);
        const uint64_t width = upper-lower;

        hipMalloc(&data_gpu[gpu], sizeof(uint64_t)*width);
    }    
    timer.stop("allocate memory");
    check_last_error();

    timer.start();
    encrypt_cpu(data_cpu, num_entries, num_iters, openmp);
    timer.stop("encrypt data on CPU");

    overall.start();
    timer.start();
    for (uint64_t gpu = 0; gpu < num_gpus; gpu++) {

        hipSetDevice(gpu);

        const uint64_t lower = chunk_size*gpu;
        const uint64_t upper = min(lower+chunk_size, num_entries);
        const uint64_t width = upper-lower;

        hipMemcpy(data_gpu[gpu], data_cpu+lower, 
               sizeof(uint64_t)*width, hipMemcpyHostToDevice);
    }
    timer.stop("copy data from CPU to GPU");
    check_last_error();

    timer.start();
    for (uint64_t gpu = 0; gpu < num_gpus; gpu++) {

        hipSetDevice(gpu);

        const uint64_t lower = chunk_size*gpu;
        const uint64_t upper = min(lower+chunk_size, num_entries);
        const uint64_t width = upper-lower;
        
        decrypt_gpu<<<80*32, 64>>>(data_gpu[gpu], width, num_iters);
    }
    timer.stop("decrypt data on the GPU");
    check_last_error();

    timer.start();
    for (uint64_t gpu = 0; gpu < num_gpus; gpu++) {

        hipSetDevice(gpu);

        const uint64_t lower = chunk_size*gpu;
        const uint64_t upper = min(lower+chunk_size, num_entries);
        const uint64_t width = upper-lower;

        hipMemcpy(data_cpu+lower, data_gpu[gpu], 
                   sizeof(uint64_t)*width, hipMemcpyDeviceToHost);
    }
    timer.stop("copy data from GPU to CPU");
    overall.stop("total time on GPU");
    check_last_error();

    timer.start();
    const bool success = check_result_cpu(data_cpu, num_entries, openmp);
    std::cout << "STATUS: test " 
              << ( success ? "passed" : "failed")
              << std::endl;
    timer.stop("checking result on CPU");

    timer.start();
    hipHostFree(data_cpu);
    for (uint64_t gpu = 0; gpu < num_gpus; gpu++) {

        hipSetDevice(gpu);
        hipFree(data_gpu[gpu]);
    }
    timer.stop("free memory");
    check_last_error();
}
